#include <stdio.h>
#include <stdlib.h>

#include "operaciones.h"

int multiplicarVectores(int* v1, int* v2, int size)
{
	int resultado = 0;
	
	for (int i = 0; i < size; ++i)
	{
		resultado += v1[i] * v2[i];
	}
	
	return resultado;
}

void multiplicarMatrices(int* m1, int* m2, int* mRes, int numFilas, int numColumnas)
{
	for(int i = 0; i < numFilas; ++i)
		for(int j = 0; j < numColumnas; ++j)
			mRes[i * numFilas + j] = multiplicarVectores(&(m1[i * numFilas]), 
                                                                    &(m2[j * numColumnas]),
			                                            numColumnas);
}
