#include <stdio.h>
#include <stdlib.h>
#include <mpi.h>

#include "operaciones.h"


#define TAG_DATO 0
#define TAG_OPERACION 1

#define OP_ADD 0
#define OP_MUL 1

void master(int argc, char** argv, int rank, int nproc)
{
	int numFilas = 100;
	int numColumnas = 100;
	int mat1[numFilas][numColumnas];
	int mat2[numFilas][numColumnas];
	int matRes[numFilas][numColumnas];

	int operacion = OP_MUL;

	MPI_Status status;
	
	//Inicializa las matrices
	for(int i = 0; i < 100; ++i)
	{
		for(int j = 0; j < 100; ++j)
		{
			mat1[i][j] = 1;
			mat2[i][j] = 1;
		}
	}

	// Mensajes pidiendo operaciones
	for(int slave = 1; slave < nproc; ++slave)
	{
		MPI_Send(&numFilas, 1, MPI_INT, slave, TAG_DATO, MPI_COMM_WORLD);
		MPI_Send(&numColumnas, 1, MPI_INT, slave, TAG_DATO, MPI_COMM_WORLD);
		MPI_Send(&(mat1[0][0]), numFilas * numColumnas, MPI_INT, slave, TAG_DATO, MPI_COMM_WORLD);
		MPI_Send(&(mat2[0][0]), numFilas * numColumnas, MPI_INT, slave, TAG_DATO, MPI_COMM_WORLD);
		
		// Solicitar operacion
		MPI_Send(&operacion, 1, MPI_INT, slave, TAG_OPERACION, MPI_COMM_WORLD);
	}

	// Mensajes Recibiendo Resultados
	for(int slave = 1; slave < nproc; ++slave)
	{
		MPI_Recv(&(matRes[0][0]), numFilas * numColumnas, MPI_INT, slave, 
			 TAG_DATO, MPI_COMM_WORLD, &status);
	}

	printf("MASTER: matriz multiplicada: \n");
	imprimeMatriz(&(matRes[0][0]), numFilas, numColumnas);
	
}

void esclavo(int argc, char** argv, int rank, int nproc)
{

	int master = 0;

	int numFilas;
	int numColumnas;

	int* mat1;
	int* mat2;

	int* matRes;
	
	int operacion;
	
	MPI_Status status;

	// Recibir mensajes mandados por master
	MPI_Recv(&numFilas, 1, MPI_INT, master, TAG_DATO, MPI_COMM_WORLD, &status);
	MPI_Recv(&numColumnas, 1, MPI_INT, master, TAG_DATO, MPI_COMM_WORLD, &status);


	mat1 = (int*)malloc(numFilas * numColumnas * sizeof(int));
	mat2 = (int*)malloc(numFilas * numColumnas * sizeof(int));

	// Declarar variable para guardar el resultado
	matRes = (int*)malloc(numFilas * numColumnas * sizeof(int));


	MPI_Recv(mat1, numFilas * numColumnas, MPI_INT, master, TAG_DATO, MPI_COMM_WORLD, &status);
	MPI_Recv(mat2, numFilas * numColumnas, MPI_INT, master, TAG_DATO, MPI_COMM_WORLD, &status);
	MPI_Recv(&operacion, 1, MPI_INT, master, TAG_OPERACION, MPI_COMM_WORLD, &status);
	

	switch(operacion)
	{
		case OP_ADD:
			printf("No implementada suma\n");
			break;
		case OP_MUL:
			multiplicarMatrices(mat1, mat2, matRes, numFilas, numColumnas);
			break;
		default:
			printf("No implementada suma\n");
	}

	MPI_Send(matRes, numFilas * numColumnas, MPI_INT, master, TAG_DATO, MPI_COMM_WORLD);

	char* nombreFich = (char *)malloc(100);
	sprintf(nombreFich, "/home/estudiante/Eugenio.Salguero/bin/%dmatrixoutEugenio.bin", rank);
	FILE* saved = fopen(nombreFich, "w");
	fwrite(matRes, numFilas * numColumnas, sizeof(int), saved);
	fclose(saved);

}

int main(int argc, char** argv)
{
	// en funcion del rank se pasa a ser maestro o esclavo
	int rank;
	int nproc;

	MPI_Init(&argc, &argv);
	MPI_Comm_size(MPI_COMM_WORLD, &nproc);
	MPI_Comm_rank(MPI_COMM_WORLD, &rank);
	
	switch(rank)
	{
		case 0: master(argc, argv, rank, nproc);
			break;
		
		default: esclavo(argc, argv, rank, nproc);
			break;
		
	}


	MPI_Finalize();

	return 0;
}
